#include "hip/hip_runtime.h"
#include "basic/math_function.h"
#include ""

namespace surfing
{
	void surfing_gpu_memcpy(const size_t N, const void* X, void* Y)
	{
		if (X != Y)
		{
			CUDA_CHECK(hipMemcpy(Y, X, N, hipMemcpyDefault));
		}
	}

	template<>
	void surfing_gpu_asum<float>(hipblasHandle_t handle, int N, const float *X, int incx, float *result)
	{
		CUBLAS_CHECK(hipblasSasum(handle, N, X, incx, result));
	}
	template<>
	void surfing_gpu_asum<double>(hipblasHandle_t handle, int N, const double *X, int incx, double *result)
	{
		CUBLAS_CHECK(hipblasDasum(handle, N, X, incx, result));
	}

	template<>
	void surfing_gpu_nrm2<float>(hipblasHandle_t handle, int N, const float *X, int incx, float *result)
	{
		CUBLAS_CHECK(hipblasSnrm2(handle, N, X, incx, result));
	}
	template<>
	void surfing_gpu_nrm2<double>(hipblasHandle_t handle, int N, const double *X, int incx, double *result)
	{
		CUBLAS_CHECK(hipblasDnrm2(handle, N, X, incx, result));
	}

	template<>
	void surfing_gpu_dot<float>(hipblasHandle_t handle, int N, const float *X, int incx, const float* Y, int incy, float *result)
	{
		CUBLAS_CHECK(hipblasSdot(handle, N, X, incx,Y,incy,result));
	}
	template<>
	void surfing_gpu_dot<double>(hipblasHandle_t handle, int N, const double *X, int incx, const double* Y, int incy, double *result)
	{
		CUBLAS_CHECK(hipblasDdot(handle, N, X, incx, Y, incy, result));
	}


	template<>
	void surfing_gpu_gemm<float>(hipblasHandle_t handle, hipblasOperation_t transa,
		hipblasOperation_t transb, int m, int n, int k,
		const float *alpha, const float *A, int lda,
		const float *B, int ldb,
		const float *beta, float *C, int ldc)
	{

		CUBLAS_CHECK(hipblasSgemm(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc));

	}
	template<>
	void surfing_gpu_gemm<double>(hipblasHandle_t handle, hipblasOperation_t transa,
		hipblasOperation_t transb, int m, int n, int k,
		const double *alpha, const double *A, int lda,
		const double *B, int ldb,
		const double *beta, double *C, int ldc)
	{
		CUBLAS_CHECK(hipblasDgemm(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc));
	}

	template<>
	void surfing_gpu_geam<float>(hipblasHandle_t handle, hipblasOperation_t transa,
		hipblasOperation_t transb, int m, int n,
		const float *alpha, const float *A, int lda,
		const float *beta, const float *B, int ldb,
		float *C, int ldc)
	{
		CUBLAS_CHECK(hipblasSgeam(handle, transa, transb, m, n, alpha, A, lda, beta, B, ldb, C, ldc));
	}
	template<>
	void surfing_gpu_geam<double>(hipblasHandle_t handle, hipblasOperation_t transa,
		hipblasOperation_t transb, int m, int n,
		const double *alpha, const double *A, int lda,
		const double *beta, const double *B, int ldb,
		double *C, int ldc)
	{
		CUBLAS_CHECK(hipblasDgeam(handle, transa, transb, m, n, alpha, A, lda, beta, B, ldb, C, ldc));
	}


	template <>
	void surfing_gpu_gemv<float>(hipblasHandle_t handle, hipblasOperation_t trans,
		int m, int n, const float* alpha,
		const float* A, int lda,
		const float* x, int incx,
		const float* beta, float* y, int incy)
	{
		CUBLAS_CHECK(hipblasSgemv(handle, trans, m, n, alpha, A, lda, x, incx, beta, y, incy));
	}
	template <>
	void surfing_gpu_gemv<double>(hipblasHandle_t handle, hipblasOperation_t trans,
		int m, int n, const double* alpha,
		const double* A, int lda,
		const double* x, int incx,
		const double* beta, double* y, int incy)
	{
		CUBLAS_CHECK(hipblasDgemv(handle, trans, m, n, alpha, A, lda, x, incx, beta, y, incy));
	}

	template<>
	void surfing_gpu_max<float>(hipblasHandle_t handle, int n, const float* X, int incx, int* result)
	{
		CUBLAS_CHECK(hipblasIsamax(handle, n, X, incx, result));
	}
	template<>
	void surfing_gpu_max<double>(hipblasHandle_t handle, int n, const double * X, int incx, int* result)
	{
		CUBLAS_CHECK(hipblasIdamax(handle, n, X, incx, result));
	}

	template<>
	void surfing_gpu_axpy<float>(hipblasHandle_t handle, int N, const float* alpha,
		const float* X, int incx, float* Y, int incy)
	{
		CUBLAS_CHECK(hipblasSaxpy(handle, N, alpha, X, incx, Y, incy));
	}
	template<>
	void surfing_gpu_axpy<double>(hipblasHandle_t handle, int N, const double* alpha,
		const double* X, int incx, double* Y, int incy)
	{
		CUBLAS_CHECK(hipblasDaxpy(handle, N, alpha, X, incx, Y, incy));
	}

	template<>
	void surfing_gpu_scal<float>(hipblasHandle_t handle, int N, const float* alpha, float* X, int incx)
	{
		CUBLAS_CHECK(hipblasSscal(handle, N, alpha, X, incx));
	}
	template<>
	void surfing_gpu_scal<double>(hipblasHandle_t handle, int N, const double* alpha, double* X, int incx)
	{
		CUBLAS_CHECK(hipblasDscal(handle, N, alpha, X, incx));
	}

	template<>
	void surfing_gpu_axpby<float>(hipblasHandle_t handle, int N, const float* alpha, const float* X, const float* beta, float* Y)
	{
		surfing_gpu_scal<float>(handle, N, beta, Y, 1);
		surfing_gpu_axpy<float>(handle, N, alpha, X, 1, Y, 1);
	}
	template<>
	void surfing_gpu_axpby <double>(hipblasHandle_t handle, int N, const double* alpha, const double* X, const double* beta, double* Y)
	{
		surfing_gpu_scal<double>(handle, N, beta, Y, 1);
		surfing_gpu_axpy<double>(handle, N, alpha, X, 1, Y, 1);
	}

	template <typename Dtype>
	__global__ void set_kernel(const int N, const Dtype alpha, Dtype* X)
	{
		CUDA_KERNEL_LOOP(index, N)
		{
			X[index] = alpha;
		}
	}
	template <> 
	void surfing_gpu_set<float>(const int N, const float alpha, float *X)
	{
		set_kernel<float> <<<SURFING_GET_BLOCK(N), SURFING_CUDA_NUM_THREADS >>>(N, alpha, X);
	}
	template <> 
	void surfing_gpu_set(const int N, const double alpha, double *X)
	{
		set_kernel<double> <<<SURFING_GET_BLOCK(N), SURFING_CUDA_NUM_THREADS >>>(N, alpha, X);
	}

	template <typename Dtype>
	__global__ void add_scalar_kernel(const int N, const Dtype alpha, Dtype* X)
	{
		CUDA_KERNEL_LOOP(index, N)
		{
			X[index] += alpha;
		}
	}
	template <>
	void surfing_gpu_add_scalar<float>(const int N, const float alpha, float *X)
	{
		add_scalar_kernel<float> <<<SURFING_GET_BLOCK(N), SURFING_CUDA_NUM_THREADS >>>(N, alpha, X);
	}
	template <>
	void surfing_gpu_add_scalar<double>(const int N, const double alpha, double *X)
	{
		add_scalar_kernel<double> <<<SURFING_GET_BLOCK(N), SURFING_CUDA_NUM_THREADS >>>(N, alpha, X);
	}

	__global__ void rounding_kernel(const int N, unsigned int range, unsigned int * X)
	{
		CUDA_KERNEL_LOOP(index, N)
		{
			X[index] %= range;
		}
	}
	void surfing_gpu_rounding(const int N, unsigned int range, unsigned int * X)
	{
		rounding_kernel<<<SURFING_GET_BLOCK(N), SURFING_CUDA_NUM_THREADS >>>(N, range, X);
	}
}

